
#include <hip/hip_runtime.h>
__global__ void kernel(int *a) {
    int i = gridDim.x;
    a[i] = i;
}

int main(void) {

    int *a;
    hipMalloc(&a, 4);

    kernel<<<1,1>>>(a);
    hipError_t err = hipDeviceSynchronize();
    if (hipSuccess != err) {
        return 1;
    }
    return 0;
}