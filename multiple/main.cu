#include "kernel.h"

int main(void) {

    float *a;
    hipMalloc(&a, 4);

    kernel<<<1,1>>>(a);
    hipError_t err = hipDeviceSynchronize();
    if (hipSuccess != err) {
        return 1;
    }
    return 0;
}